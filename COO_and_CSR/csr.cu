#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <device_launch_parameter.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <random>

// Function to generate adjacency matrix
int init_adjacency_matrix(int *adj, int seqLength, int embDimension, int sparcity)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);

    int nnz = 0;
    // Generate and print the matrix
    for (int i = 0; i < seqLength; ++i)
    {
        for (int j = 0; j < embDimension; ++j)
        {
            // Generate a random number and compare with sparsity
            adj[i * embDimension + j] = dis(gen) < sparsity ? 1 : 0;
            if (adj[i * embDimension + j] == 1)
            {
                nnz++;
            }
        }
    }
    return nnz;
}

// Function for generating random matrix using Xavier(Glorot) initialization

// Generate random matrix of values between 0 to 99
void init_matrix(int *mat, int seqLength, int embDimension)
{
    for (int i = 0; i < seqLength; i++)
    {
        for (int j = 0; j < embDimension; j++)
        {
            mat[i * embDimension + j] = rand() % 100;
        }
    }
}

// Kernels
// 1st matrix multiplication
__global__ void QxK_and_exp(float *Q, float *K, int seqLength, int embDimension, int *row, int *col, float *w, int nopt, float *exp_sum)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < seqLength)
    {
        float sum = 0.0f;

        // Iterate through non-zero elements for the row corresponding to tid
        for (int idx = row[tid]; idx < row[tid + 1]; ++idx)
        {
            int colIndex = col[idx]; // Get the column index
            float value = w[idx];    // Get the corresponding weight

            // Calculate dot product
            float dot_product = 0.0f;
            for (int d = 0; d < embDimension; ++d)
            {
                dot_product += Q[tid * embDimension + d] * K[colIndex * embDimension + d];
            }

            // Exponential of the dot product
            float exp_value = exp(dot_product) * value;
            sum += exp_value; // Sum up the exponential values
        }

        // Store the sum of exponentials for this row
        exp_sum[tid] = sum;
    }
}

// Divide the values in w matrix with exp_sum
__global__ void sum_div(float *w, float *exp_sum, int *row, int nopt)
{
    // Identifying the corresponding part from the coo
    int initial = ((blockIdx.x * blockDim.x) + threadIdx.x) * nopt;
    for (int i = initial; i < (initial + nopt); i++)
    {
        w[i] /= exp_sum[row[i]];
    }
}

// Kernel for sparse matrix (COO) * dense matrix multiplication with reduction
__global__ void wxV(int *d_row, int *d_col, float *d_w, int nnz, float *d_V, int seqLength, int embDimension, float *d_result, int ncpb)
{

    extern __shared__ float sharedMemory[]; // Dynamically allocated shared memory

    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread is responsible for computing one element in the result
    for (int i = 0; i < ncpb; ++i)
    {
        int idx = globalIdx * ncpb + i; // Compute the current index for this thread

        // Ensure we are within the valid range
        if (idx < seqLength)
        {
            float value = 0.0f;

            // Access the CSR format
            int row_start = d_row[idx];   // Start index of the row
            int row_end = d_row[idx + 1]; // End index of the row

            // Iterate over non-zero elements in the current row
            for (int j = row_start; j < row_end; ++j)
            {
                int colIndex = d_col[j]; // Column index
                float weight = d_w[j];   // Weight for the non-zero element

                // Multiply by corresponding element in V
                for (int k = 0; k < embDimension; ++k)
                {
                    value += weight * d_V[colIndex * embDimension + k];
                }
            }

            // Store result in shared memory
            sharedMemory[threadIdx.x] = value;
            __syncthreads();

            // Perform reduction in shared memory
            for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
            {
                if (threadIdx.x < stride)
                {
                    sharedMemory[threadIdx.x] += sharedMemory[threadIdx.x + stride];
                }
                __syncthreads();
            }

            // The final result for this block's thread
            if (threadIdx.x == 0)
            {
                atomicAdd(&d_result[idx], sharedMemory[0]); // Write result to global memory
            }
        }
    }
}

int main()
{
    // size of matrices
    int seqLength = 1 << 4;    // Sequence length
    int embDimension = 1 << 4; // Embedding dimension

    // Sparcity of adjacency matrix - Sparcity of the Attention
    int sparcity = 0.5;

    // define size  of matrices - for memory allocation
    size_t matrixSize = sizeof(float) * seqLenght * embDimension;     // for Q K V and result matrices
    size_t attentionMatrixSize = sizeof(int) * seqLength * seqLenght; // for adj
    size_t sumOfVectorsSize = sizeof(float) * seqLength;              // for sumOfVectors

    // Note: prefix h for a variable represents that the momory allocation is on host and prefix d for a variable represents that the memory allocation is on device

    // Initialize host pointers
    float *h_Q, *h_K, *h_V, *h_sumOfVectors, *h_result;
    int *h_attentionMatrix;

    // Allocate memory at the host
    h_Q = (*float)malloc(matrixSize);
    h_K = (*float)malloc(matrixSize);
    h_V = (*float)malloc(matrixSize);
    h_attentionMatrix = (*int)malloc(attentionMatrixSize);
    h_sumOfVectors = (*float)malloc(sumOfVectorsSize);
    h_result = (*float)malloc(matrixSize);

    // The memory allocation for attentionMatrix on device is not needed as the coo matrix is being computed on host

    // Initialize device pointers
    float *d_Q, *d_K, *d_V, *d_sumOfVectors, *d_result;

    // allocate memory on the device
    hipMalloc(&d_Q, matrixSize);
    hipMalloc(&d_K, matrixSize);
    hipMalloc(&d_V, matrixSize);
    hipMalloc(&d_sumOfVectors, sumOfVectorsSize);
    hipMalloc(&d_result, matrixSize);

    // Initialize matrices with random values
    init_matrix(h_Q, seqLength, embDimension);
    init_matrix(h_K, seqLength, embDimension);
    init_matrix(h_V, seqLength, embDimension);
    int nnz = init_adjacency_matrix(h_adj, seqLength, embDimension, sparcity);

    // copy memory to the device
    hipMemcpy(d_Q, h_Q, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, matrixSize, hipMemcpyHostToDevice);

    // represent adjacency matrix in coo form
    int *h_row, *h_col;
    float *h_w; // for storing intermediate result
    size_t nonZeroInt_Col = nnz * sizeof(int);
    size_t nonZeroInt_Row = seqLength * sizeof(int);
    size_t nonZeroFloat = nnz * sizeof(float);
    h_row = (*int)malloc(nonZeroInt_Row);
    h_col = (*int)malloc(nonZeroInt_Col);
    h_w = (*float)malloc(nonZeroFloat);

    int *d_row, *d_col;
    float *d_w;
    hipMalloc(&d_row, nonZeroInt_Row);
    hipMalloc(&d_col, nonZeroInr_Col);
    hipMalloc(&d_w, nonZeroFloat);

    // Generate COO
    int cooIndex = 0; // index for coo
    for (int i = 0; i < seqLength; i++)
    {
        for (int j = 0; j < embDimension; j++)
        {
            if (h_adj[i * embDimension + j] == 1)
            {
                h_col[cooIndex] = j;
                cooIndex++; // increment after each non-zero element
            }
        }
        h_row[i] = cooIndex - 1
    }

    free(h_adj);

    // transfer coo representation of adj
    hipMemcpy(d_row, h_row, nonZeroInt, hipMemcpyHostToDevice);
    hipMemcpy(d_col, h_col, nonZeroInt, hipMemcpyHostToDevice);

    // Defining required GPU architecture

    // Define BlockSize
    int BlockSize = 256;

    // Calculate gridSize
    int GridSize = 16;

    // Number of operations per thread
    int nopt = ceil(nnz / (BlockSize * GridSize));

    // Initialize allocated memory to zero
    memset(h_w, 0, nonZeroFloat);
    hipMemcpy(d_w, h_w, nonZeroFloat, hipMemcpyHostToDevice);

    // Initialize h_sum to 0 and copy that to device meomory
    memset(h_sumOfVectors, 0, sumOfVectorsSize);
    hipMemcpy(d_sumOfVectors, h_sumOfVectors, sumOfVectorsSize, hipMemcpyHostToDevice);

    // Call the Kernel (for first matrix multiplication and also exponential and addition of values of corresponding rows)
    QxK_and_exp<<<GridSize, BlockSize>>>(d_Q, d_K, seqLength, embDimension, d_row, d_col, d_w, nopt, d_sumOfVectors);

    hipDeviceSynchronize();

    // call the kernel (for division by sum of exponentials)
    sum_div<<<GridSize, BlockSize>>>(d_w, d_sumOfVectors, d_row, nopt);

    hipDeviceSynchronize();

    // Initialize result matrix to zero and pass it to device memory
    memset(h_result, 0, matrixSize);
    hipMemcpy(d_result, h_result, matrixSize, hipMemcpyHostToDevice);

    // call the kernel (for final matrix multiplication) - method 1
    int ncpt = ceil((seqLength * embDimension) / GridSize); // number of computations per block
    wxV<<<GridSize, BlockSize>>>(d_row, d_col, d_w, nnz, d_V, seqLength, embDimension, d_result, ncpt);

    // // call the kernel (for final matrix multiplication) - method 2
    // wxV<<<GridSize, BlockSize>>>(d_w, d_row, d_col, d_V, seqLength, embDimension, d_result, nopt);

    hipDeviceSynchronize();

    // free device memory
    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_sumOfVectors);
    hipFree(d_col);
    hipFree(d_row);

    // free host memory
    free(h_Q);
    free(h_K);
    free(h_V);
    free(h_sumOfVectors);
    free(h_col);
    free(h_row);
}
