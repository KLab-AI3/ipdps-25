#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <cmath>
#include <stdexcept>
#include <iostream>
#include <stdio.h>

#include <torch/extension.h>


// The sparse-FlashAttention (BSR mask) forward CUDA kernel.
template <typename scalar_t>
__global__ void spfa_bsr_cuda_forward_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> K,
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> V,
    const torch::PackedTensorAccessor64<unsigned long int, 1, torch::RestrictPtrTraits> W_block_row_off,
    const torch::PackedTensorAccessor64<unsigned int, 1, torch::RestrictPtrTraits> W_block_col_ind,
    torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> W_val,
    torch::PackedTensorAccessor64<scalar_t, 1, torch::RestrictPtrTraits> m,
    torch::PackedTensorAccessor64<scalar_t, 1, torch::RestrictPtrTraits> l,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> O,
    int block_length,
    int maskElePerIter
) {
  // Keep track of the lower and upper bound indices of the W block column index vector.
  int lowInd = W_block_row_off[blockIdx.x];
  int upInd = W_block_row_off[blockIdx.x + 1];

  // Calculate the total number of non-masked elements in the row of the mask.
  int totNonMask = block_length * (upInd - lowInd);

  // Track the number of entries that have been calculated. This will help us figure out which block and item we are working 
  // on if more than one iteration is required to calculate the row of O.
  int calced_tracker = 0;

  // Calculate the number of times the amount of threads will fit into the embedded dimension as well as the remainder.
  int intQuotient_ED = (int)(Q.size(1) / blockDim.x);
  int remainder_ED = (Q.size(1) % blockDim.x) - 1;

  // If the thread index is less than the remainder, then increment its quotient so it reads one more value.
  if (threadIdx.x <= remainder_ED) {
    intQuotient_ED += 1;
  }

  // If the row is fully masked, then fill the output with NaN's and end the iteration.
  // NOTE: Check if this is correct or if I should set it to another value.
  if (totNonMask == 0) {
    for (int i = 0; i < intQuotient_ED; i++) {
      O[blockIdx.x][(blockDim.x * i) + threadIdx.x] = (scalar_t)NAN;
      // __syncthreads();
    }

    return;
  }

  // If the total number of non-masked elements is under the max number of mask elements per iteration, 
  // then reduce the max number of elements for this iteration.
  if (totNonMask < maskElePerIter) {
    maskElePerIter = totNonMask;
  }

  // Create the shared memory object, aligned to the size of the template data type.
  extern __shared__ __align__(sizeof(scalar_t)) unsigned char sharedMem[];

  // Get pointers to the shared memory and reinterpret the pointer's data type.
  scalar_t* Q_shared = reinterpret_cast<scalar_t*>(sharedMem);
  scalar_t* O_shared = (scalar_t*)&Q_shared[Q.size(1)];
  scalar_t* W_val_shared = (scalar_t*)&O_shared[O.size(1)];
  scalar_t* K_shared = (scalar_t*)&W_val_shared[maskElePerIter];
  scalar_t* V_shared = (scalar_t*)&K_shared[maskElePerIter * K.size(1)];
  scalar_t* m_i = (scalar_t*)&V_shared[maskElePerIter * V.size(1)];
  scalar_t* m_ij = (scalar_t*)&m_i[1];
  scalar_t* m_new_i = (scalar_t*)&m_ij[1];
  scalar_t* l_i = (scalar_t*)&m_new_i[1];
  scalar_t* l_ij = (scalar_t*)&l_i[1];
  scalar_t* l_new_i = (scalar_t*)&l_ij[1];

  // Bring Q and O into shared memory.
  for (int i = 0; i < intQuotient_ED; i++) {
    Q_shared[(blockDim.x * i) + threadIdx.x] = Q[blockIdx.x][(blockDim.x * i) + threadIdx.x];
    O_shared[(blockDim.x * i) + threadIdx.x] = O[blockIdx.x][(blockDim.x * i) + threadIdx.x];
    // __syncthreads();
  }

  // Move the softmax statistics into shared memory.
  //// - m is initialized with 0.0.
  //// - l is initialized with -inf.
  //// - If they are set to something else in shared memory, then those values will be used.
  if (threadIdx.x == 0) {
    m_i[0] = m[blockIdx.x];
    l_i[0] = l[blockIdx.x];
  }

  // Block until all threads are synchronized after the last operation.
  __syncthreads();

  // Calculate the number of iterations that the block will have to perform.
  int blockIters = ceil(((float)totNonMask / (float)maskElePerIter));

  // Calculate the number of masked terms that remain on the last iteration.
  int numLastIter = (totNonMask % maskElePerIter);

  // The main calculation loop.
  for (int b_i = 0; b_i < blockIters; b_i++) {
    // If we're on the last iteration and it's not the first, then set the number of masked elements equal to those 
    // that have not been visited yet.
    if ((b_i == (blockIters - 1)) && (b_i != 0)) {
      maskElePerIter = numLastIter;
    }

    // Calculate the column indices and bring K and V into shared memory.
    //// - W's value vector is overwritten the first iteration, so there is no need to read from HBM or initialize it to something.
    //// - K_shared is contiguous along the rows of K.
    //// - V_shared is contiguous along the columns of V.
    if (threadIdx.x < maskElePerIter) {
      // Find where the thread's position is within the total number of calculations that must be conducted.
      int position = calced_tracker + threadIdx.x;

      // Find which block index the thread is associated with and adjust it using the lower bound index pointer.
      int block_ind = floor(position / block_length) + lowInd;

      // Find the thread's position within said block.
      int block_in_ind = position % block_length;

      // Calculate and store the column index of the mask so we don't have to do it each iteration below.
      int col_ind = (block_length * W_block_col_ind[block_ind]) + block_in_ind;

      for (int i = 0; i < K.size(1); i++) {
        K_shared[(K.size(1) * threadIdx.x) + i] = K[col_ind][i];
        V_shared[(maskElePerIter * i) + threadIdx.x] = V[col_ind][i];
      }
    }

    // Update the tracker to know how many elements of the mask have been operated on thus far.
    calced_tracker += maskElePerIter;

    __syncthreads();

    if (threadIdx.x < maskElePerIter) {
      scalar_t acc = 0.0;

      for (int i = 0; i < Q.size(1); i++) {
        acc += Q_shared[i] * K_shared[(Q.size(1) * threadIdx.x) + i];
      }

      W_val_shared[threadIdx.x] = acc / sqrt((float)Q.size(1));
    }

    __syncthreads();

    if (threadIdx.x == 0) {
      scalar_t row_max = - (1.0 / 0.0);

      for (int i = 0; i < maskElePerIter; i++) {
        if (W_val_shared[i] > row_max) {
          row_max = W_val_shared[i];
        }
      }

      m_ij[0] = row_max;
    }

    __syncthreads();

    if (threadIdx.x < maskElePerIter) {
      W_val_shared[threadIdx.x] = exp(W_val_shared[threadIdx.x] - m_ij[0]);
    }

    __syncthreads();

    if (threadIdx.x == 0) {
      scalar_t row_sum = 0.0;

      for (int i = 0; i < maskElePerIter; i++) {
        row_sum += W_val_shared[i];
      }

      l_ij[0] = row_sum;

      if (m_i[0] > m_ij[0]) {
        m_new_i[0] = m_i[0];
      }
      else {
        m_new_i[0] = m_ij[0];
      }

      l_new_i[0] = (l_i[0] * exp(m_i[0] - m_new_i[0])) + (l_ij[0] * exp(m_ij[0] - m_new_i[0]));
    }

    __syncthreads();

    // NOTE: This does not work for sizes greater than number of threads.
    if (threadIdx.x < V.size(1)) {
      scalar_t acc = 0.0;

      for (int i = 0; i < maskElePerIter; i++) {
        acc += W_val_shared[i] * V_shared[(threadIdx.x * maskElePerIter) + i];
      }

      O_shared[threadIdx.x] = (1.0 / l_new_i[0]) * 
        ((l_i[0] * O_shared[threadIdx.x] * exp(m_i[0] - m_new_i[0])) + 
        (acc * exp(m_ij[0] - m_new_i[0])));
    }

    __syncthreads();

    if (threadIdx.x == 0) {
      m_i[0] = m_new_i[0];
      l_i[0] = l_new_i[0];
    }

    __syncthreads();

  }

  // Update the softmax statistics in HBM.
  if (threadIdx.x == 0) {
      m[blockIdx.x] = m_new_i[0];
      l[blockIdx.x] = l_new_i[0];
  }

  __syncthreads();

  for (int i = 0; i < intQuotient_ED; i++) {
    O[blockIdx.x][(blockDim.x * i) + threadIdx.x] = O_shared[(blockDim.x * i) + threadIdx.x];
    // __syncthreads();
  }
  
}


// The CUDA kernel dispatch interface for the sparse-FlashAttention (BSR mask) forward CUDA kernel.
torch::Tensor spfa_bsr_cuda_forward(
    torch::Tensor Q,
    torch::Tensor K,
    torch::Tensor V,
    torch::Tensor W_block_row_off,
    torch::Tensor W_block_col_ind,
    torch::Tensor W_val,
    torch::Tensor m,
    torch::Tensor l,
    torch::Tensor O,
    int block_length
) {
  // Initialize the variables to hold the desired device attributes.
  //// - Max number of threads per block.
  //// - Max shared memory per block (bytes).
  int maxThrPB, maxSMemPB;

  // Get the device attributes.
  hipDeviceGetAttribute(&maxThrPB, hipDeviceAttributeMaxThreadsPerBlock, 0);
  hipDeviceGetAttribute(&maxSMemPB, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

  // Set the grid dimensions for the problem.
  //// - The number of threads is set to the max per block for the device.
  //// - The number of blocks is equal to the number of rows in Q.
  const int threads = maxThrPB;
  const dim3 blocks(Q.size(0));

  // Store the embedded dimension.
  int embD = Q.size(1);

  // Calculate the max number of mask elements that will be utilized for each iteration of building O.
  //// - maxSMemPB -> The amount of shared memory available to a block in bytes.
  //// - (2 * embD * sizeof(Q.type())) -> (numerator) The amount of shared memory that storing Q and O will require.
  //// - (6 * sizeof(l.type())) -> The amount of shared memory that storing the softmax statistics will require.
  //// - sizeof(W_val.type()) -> The amount of shared memory that storing the mask's values will require.
  //// - (2 * embD * sizeof(Q.type())) -> (denominator) The amount of shared memory that storing K and V will require.
  int maskElePerIter = floor((maxSMemPB - (2 * embD * sizeof(Q.type())) - (6 * sizeof(l.type()))) / 
    (sizeof(W_val.type()) + (2 * embD * sizeof(Q.type()))));

  // If the number of mask elements per iteration is less than 1, then throw an error.
  if (maskElePerIter < 1) {
    throw std::invalid_argument( "The embedded dimension is too large for the amount of shared memory on your GPU." );
  }

  // Dispatcher that handles launching the correctly typed function from the generic implementation.
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(Q.type(), "spfa_bsr_forward_cuda", ([&] {
    spfa_bsr_cuda_forward_kernel<scalar_t><<<blocks, 256, maxSMemPB>>>(
        Q.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
        K.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
        V.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
        W_block_row_off.packed_accessor64<unsigned long int, 1, torch::RestrictPtrTraits>(),
        W_block_col_ind.packed_accessor64<unsigned int, 1, torch::RestrictPtrTraits>(),
        W_val.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
        m.packed_accessor64<scalar_t, 1, torch::RestrictPtrTraits>(),
        l.packed_accessor64<scalar_t, 1, torch::RestrictPtrTraits>(),
        O.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
        block_length,
        maskElePerIter
      );
  }));

  // Return the output tensor.
  return O;
}
